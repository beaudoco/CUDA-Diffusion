// compute.cu
//
// driver and kernel call


#include <hip/hip_runtime.h>
#include <stdio.h>

#define THREADS_PER_BLOCK 32
 
__global__ void compute_d (float *c_d, int arrSize, int timeStep, int timeSteps)
{
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    if (x <= arrSize) {
        if (x % 2 == timeStep % 2 && x <= timeStep)
        {
            if (timeStep > timeSteps && x <= (timeStep - timeSteps - arrSize))
            {
                
            } else 
            {
                if (x == 0)
                {
                    c_d[x] = (100.0 + c_d[x + 1]) / 2.0;
                } else if (x == arrSize - 1)
                {
                    c_d[x] = (c_d[x - 1] + c_d[x]) / 2.0;
                } else
                {
                    c_d[x] = (c_d[x - 1] + c_d[x + 1]) / 2.0;
                }
            }
        }
        __syncthreads();
	}
		
}

extern "C" void computeArr (float *metalRod, int arrSize, int timeSteps)
{
    float *c_d;
    int i = 0;
    
    hipMalloc ((void**) &c_d, sizeof(float) * arrSize);
    hipMemcpy (c_d, metalRod, sizeof(float) * arrSize, hipMemcpyHostToDevice);
    
    for (i = 0; i < (2*(timeSteps - 1)) + arrSize; i++)
    {
        compute_d <<< ceil((float) arrSize/THREADS_PER_BLOCK), THREADS_PER_BLOCK >>> (c_d, arrSize, i, timeSteps);
    }
	
	hipError_t err = hipGetLastError();
	if (err != hipSuccess)
		printf ("CUDA error: %s\n", hipGetErrorString(err));
		
	hipMemcpy (metalRod, c_d, sizeof(float) * arrSize, hipMemcpyDeviceToHost);
	hipFree (c_d);
}

